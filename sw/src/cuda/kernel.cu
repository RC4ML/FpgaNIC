#include "hip/hip_runtime.h"
#include "kernel.cuh"
#include "util.cuh"
 __global__ void verify(int * data,size_t length,int offset){
	 int count=0;
	 int count_1024=0;
	 int flag=1;
	 BEGIN_SINGLE_THREAD_DO
		size_t op_num = size_t(length/sizeof(int));
		for(int i=0;i<op_num;i++){
			if((i%16!=0)&&data[i]!=i+offset){
				if(data[i]-i-offset == 1024){
					count_1024+=1;
				}
				if(flag==1){
					printf("verify data failed! index:%d data:%d which should be %d last one is %x\n",i,data[i],i+offset,data[i-1]);
					flag=0;
				}
				count+=1;
			}
		}
		printf("wrong num:%d  count1024:%d\n",count,count_1024);
		if(flag==1){
			printf("verify data success!\n");
		}
		
	 END_SINGLE_THREAD_DO
 }

__global__ void compute(int * data,size_t length,int offset){
	int index = blockIdx.x*blockDim.x+threadIdx.x;	
	int total_threads = gridDim.x*blockDim.x;
	// BEGIN_SINGLE_THREAD_DO
	// 	printf("total_threads:%d\n",total_threads);
	// END_SINGLE_THREAD_DO
	size_t op_num = size_t(length/sizeof(int));
	int iter_num = int(op_num/total_threads);

	for(int i=0;i<iter_num;i++){
		data[i*total_threads+index]	=	i*total_threads+index+offset;
	}
	
	BEGIN_SINGLE_THREAD_DO
		// for(int i=0;i<16;i++){
		// 	printf("%d ",data[i]);
		// }
		// printf("\n");
		printf("---function compute done!\n");
		// for(int i=0;i<64;i++){
		// 	printf("%d  ",data[i]);
		// }
		// printf("\n");
	END_SINGLE_THREAD_DO
}

__global__ void movThread(param_cuda_thread_t param){
	//printf("readCount:%d 	writeCount:%d\n",devReadCountAddr[0],devWriteCountAddr[0]);
	int rCount;
	int moveCount=0;
	unsigned int* dataAddr = param.devVAddr0 + int(2*1024*1024/4);
	int step = int(2*1024*1024/4);
	int index = blockIdx.x*blockDim.x+threadIdx.x;
	int pagesToRead = int(param.data_length/2/1024/1024);

	rCount = param.devReadCountAddr0[0];
	if(0 == index){
		printf("index:%d rcount:%d devReadCountAddr0: %d   devWriteCountAddr: %d\n",index,rCount,param.devReadCountAddr0[0],param.devWriteCountAddr0[0]);
		printf("index:%d blocks:%d threadsPerBlock:%d pages to read:%d\n",index,param.blocks,param.threadsPerBlock,pagesToRead);
	}
	int tmp = param.devReadCountAddr0[0];
	
	{//ptx test
		
		if(index==1){
			uint64_t volatile addr = (uint64_t)(dataAddr+16*5+16*10*index);
			int volatile y=0;
			uint64_t s=0;
			uint64_t e=0;
			asm volatile(
				"mov.u64 %0,%%clock64;\n\t"
				"ld.u32 %2,[%3];\n\t"//没有就是7 有的话16   ca=16  cg=16   cs=16 lu=16  cv=16
				"mov.u64 %1,%%clock64;\n\t"
				:"=l"(s),"=l"(e),"=r"(y):"l"(addr): "memory"
			);
			printf("s:%ld e:%ld e-s:%ld y:%d\n",s,e,e-s,y);
		}
	}
	__syncthreads();
	while(1){
		// if(index==0){
		// 	param.devReadCountAddr0[0] =param.devWriteCountAddr0[0];
		// 	if(param.devReadCountAddr0[0]==(tmp+1000)){
		// 		break;
		// 	}
		// }
		
		while((index%(blockDim.x))==0 && rCount==param.devWriteCountAddr0[0]){
			continue;
		}
		__syncthreads();
		unsigned int* startDstAddr = param.dstAddr0+step*(rCount%(param.buffer_pages));
		unsigned int* startSrcAddr = dataAddr + step*(rCount%(param.buffer_pages));
		// if(index<10)
		// printf("i:%d %d %d %d\n",index,rCount,step*(rCount%100),moveCount);
		int length = int(step/param.threadsPerBlock/param.blocks);
		int stride = param.threadsPerBlock*param.blocks;
		for(int i=0;i<length;i++){
			startDstAddr[i*stride+index] = startSrcAddr[i*stride+index];
		}
		rCount+=1;
		__syncthreads();
		if(index==0){
			param.devReadCountAddr0[0] = rCount;
			//printf("index:%d  devReadCountAddr:%d   devWriteCountAddr:%d\n",index,param.devReadCountAddr0[0],param.devWriteCountAddr0[0]);
		}
		moveCount+=1;
		if(moveCount==pagesToRead){
			break;
		}
	}
	__syncthreads();
	{//ptx test
		
		if(index==1){
			uint64_t addr = (uint64_t)(dataAddr+16*5+16*3000*index);
			int y=0;
			uint64_t s=0;
			uint64_t e=0;
			asm volatile(
				"mov.u64 %0,%%clock64;\n\t"
				"ld.cv.u32 %2,[%3];\n\t"//没有就是7 有的话16   ca=16  cg=16   cs=16 lu=16  cv=16
				"mov.u64 %1,%%clock64;\n\t"
				:"=l"(s),"=l"(e),"=r"(y):"l"(addr): "memory"
			);
			printf("s:%ld e:%ld e-s:%ld y:%d\n",s,e,e-s,y);

		}
	}
	__syncthreads();
	{//check results
		if(index==0){
			printf("read done! %d %d \n",tmp,param.devReadCountAddr0[0]);
			for(int i=0;i<int(param.data_length/64);i++){
				if(param.dstAddr0[i*16] != i%32768+param.offset){
					printf("%dth with value:%d\n",i,param.dstAddr0[i*16]);
				}
			}
			printf("check done! %d %d \n",tmp,param.devReadCountAddr0[0]);
		}
	}
	

	// {//print data
	// 	if(index==0){
	// 		for(int i=0;i<int(param.data_length/64);i+=1024){
	// 			printf("i:%d  data:%d\n",i,param.dstAddr0[i*16]);
	// 		}
	// 	}
	// }

}




__global__ void writeBypassReg(volatile unsigned int *dev_addr,int *blocks){
	//printf("enter writeBypassReg thread with mapped dev_addr:%x\n",dev_addr);
	int index = blockIdx.x*blockDim.x+threadIdx.x;
	int stride = blocks[0]*blockDim.x;
	int num = int(1024*1024/4/stride);
	int sum=0;
	for(int i=0;i<num;i++){
		dev_addr[i*stride+index]=i;
		sum+=dev_addr[i*stride+index];
	}
	printf("%d %d %d \n",index,stride,num);
	printf("%d \n",sum);
}
__global__ void readBypassReg(volatile unsigned int *dev_addr,int *blocks){
	//printf("enter readBypassReg thread with mapped dev_addr:%x\n",dev_addr);
	int index = blockIdx.x*blockDim.x+threadIdx.x;
	int next_addr,sum;
	
	next_addr=dev_addr[1600+index];
	clock_t s = clock64();
	sum = dev_addr[next_addr+index];
	// __threadfence();
	clock_t e = clock64();
	printf("latency:%lu\n",e-s);
	printf("%d %d \n",next_addr,sum);
}
__global__ void writeReg(volatile unsigned int *dev_addr,int *blocks){
	
	printf("enter writeReg thread with addr:%x\n",dev_addr);
	int index = blockIdx.x*blockDim.x+threadIdx.x;
	int stride = blocks[0]*blockDim.x;
	int num = int(1024*1024/4/stride);
	clock_t start_clock = clock();
	// #pragma unroll 1
	int sum=0;
	for(int i=0;i<num;i++){
		dev_addr[i*stride+index]=i;
		//__syncthreads();
		//__threadfence();
		//__threadfence_block();
		//__threadfence_system();
		sum+=dev_addr[i*stride+index];
	}

	printf("%d %d %d \n",index,stride,num);
	printf("%d \n",sum);
}