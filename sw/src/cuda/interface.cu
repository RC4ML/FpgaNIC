#include "interface.cuh"
#include "kernel.cuh"
#include "network.cuh"
 


void socket_sample(param_interface_socket_t param_in){
	socket_context_t* context = get_socket_context(param_in.buffer_addr,param_in.tlb_start_addr,param_in.controller);
	int user_blocks=4;
	int user_thread_num=256;
	dim3 threads_per_block(user_thread_num,1);	
	dim3 num_blocks(user_blocks,1);	  
	int * data;
	size_t length = 4*256*1024*1024;
	hipMalloc(&data,length);

	sock_addr_t addr;
	addr.ip = param_in.ip;
	addr.mac = param_in.mac;
	addr.port = param_in.port;

	int* socket1;
	hipMalloc(&socket1,sizeof(int));
	hipStream_t stream;
	hipStreamCreate(&stream);

	create_socket<<<1,1,0,stream>>>(context,socket1);
	compute<<<1,1024,0,stream>>>(data,length,3);
	socket_send<<<1,2,0,stream>>>(context,socket1,data,1024,addr);
	// sleep(10);
	// printf("%x\n",param_in.controller.readReg());
	// printf("%x\n",param_in.controller.readReg());
	// printf("%x\n",param_in.controller.readReg());
	// printf("%x\n",param_in.controller.readReg());
	// printf("%x\n",param_in.controller.readReg());
	// printf("%x\n",param_in.controller.readReg());
	// printf("%x\n",param_in.controller.readReg());
}




void data_mover(param_mover param_mover){
	
	hipError_t err = hipHostRegister(param_mover.write_count_addr0,4,hipHostRegisterIoMemory);
	ErrCheck(err);
	err = hipHostRegister(param_mover.read_count_addr0,4,hipHostRegisterIoMemory);
	ErrCheck(err);

	err = hipHostRegister(param_mover.write_count_addr1,4,hipHostRegisterIoMemory);
	ErrCheck(err);
	err = hipHostRegister(param_mover.read_count_addr1,4,hipHostRegisterIoMemory);
	ErrCheck(err);

	err = hipHostRegister(param_mover.write_count_addr2,4,hipHostRegisterIoMemory);
	ErrCheck(err);
	err = hipHostRegister(param_mover.read_count_addr2,4,hipHostRegisterIoMemory);
	ErrCheck(err);

	err = hipHostRegister(param_mover.write_count_addr3,4,hipHostRegisterIoMemory);
	ErrCheck(err);
	err = hipHostRegister(param_mover.read_count_addr3,4,hipHostRegisterIoMemory);
	ErrCheck(err);

	param_cuda_thread_t param_cuda;

	param_cuda.data_length 		=	param_mover.data_length;
	param_cuda.offset			=	param_mover.offset;
	param_cuda.buffer_pages		=	param_mover.buffer_pages;

	hipHostGetDevicePointer((void **) &(param_cuda.devReadCountAddr0), param_mover.read_count_addr0, 0);
	hipHostGetDevicePointer((void **) &(param_cuda.devReadCountAddr1), param_mover.read_count_addr1, 0);
	hipHostGetDevicePointer((void **) &(param_cuda.devReadCountAddr2), param_mover.read_count_addr2, 0);
	hipHostGetDevicePointer((void **) &(param_cuda.devReadCountAddr3), param_mover.read_count_addr3, 0);

	hipHostGetDevicePointer((void **) &(param_cuda.devWriteCountAddr0), param_mover.write_count_addr0, 0);
	hipHostGetDevicePointer((void **) &(param_cuda.devWriteCountAddr1), param_mover.write_count_addr1, 0);
	hipHostGetDevicePointer((void **) &(param_cuda.devWriteCountAddr2), param_mover.write_count_addr2, 0);
	hipHostGetDevicePointer((void **) &(param_cuda.devWriteCountAddr3), param_mover.write_count_addr3, 0);

	param_cuda.devVAddr0 = param_mover.dev_addr0;
	param_cuda.devVAddr1 = param_mover.dev_addr1;
	param_cuda.devVAddr2 = param_mover.dev_addr2;
	param_cuda.devVAddr3 = param_mover.dev_addr3;

	int hostBlocks=1;
	int threadNum=1024;//8 128
	param_cuda.threadsPerBlock = threadNum;
	param_cuda.blocks = hostBlocks;
	
	
	dim3 threadsPerBlock(threadNum,1);	
	dim3 numBlocks(hostBlocks,1);	  

	//local copy
	// unsigned int * src;
	// hipMalloc(&(src),2000*1024*1024);
	// param_cuda.devVAddr0 = src;

	hipMalloc(&(param_cuda.dstAddr0),param_mover.data_length);
	// hipMalloc(&(param_cuda.dstAddr1),200*1024*1024);
	// hipMalloc(&(param_cuda.dstAddr2),200*1024*1024);
	// hipMalloc(&(param_cuda.dstAddr3),200*1024*1024);
	movThread<<<numBlocks,threadsPerBlock>>>(param_cuda); 
	// hipError_t cudaerr = hipDeviceSynchronize();
	// ErrCheck(cudaerr);
}

void write_bypass(void* addr){
	int peak_clk = 1;
	int device = 0;
	hipError_t e = hipDeviceGetAttribute(&peak_clk, hipDeviceAttributeClockRate, device);
	printf("%d\n",peak_clk);
	hipError_t err = hipHostRegister((void *)addr,1024*1024,hipHostRegisterIoMemory);
	if (err != hipSuccess)
        printf("kernel launch failed with error \"%s\".\n",
               hipGetErrorString(err));
	volatile unsigned int *devPtrAddr;
	hipHostGetDevicePointer((void **) &devPtrAddr, (void *) addr, 0);
	int host_blocks=1;
	dim3 threadsPerBlock(16,1);
	int *device_blocks;
	hipMalloc(&device_blocks, 1 * sizeof(int));
	hipMemcpy(device_blocks,&host_blocks,sizeof(int)*1,hipMemcpyHostToDevice);
	dim3 numBlocks(host_blocks,1);
	
	writeBypassReg<<<numBlocks,threadsPerBlock>>>(devPtrAddr,device_blocks);
	hipError_t cudaerr = hipDeviceSynchronize();
    if (cudaerr != hipSuccess)
        printf("kernel launch failed with error \"%s\".\n",
               hipGetErrorString(cudaerr));
}
void read_bypass(void* addr){
	int peak_clk = 1;
	int device = 0;
	hipError_t e = hipDeviceGetAttribute(&peak_clk, hipDeviceAttributeClockRate, device);
	printf("%d\n",peak_clk);
	hipError_t err = hipHostRegister((void *)addr,1024*1024,hipHostRegisterIoMemory);
	if (err != hipSuccess)
        printf("kernel launch failed with error \"%s\".\n",
               hipGetErrorString(err));
	volatile unsigned int *devPtrAddr;
	hipHostGetDevicePointer((void **) &devPtrAddr, (void *) addr, 0);
	int host_blocks=1;
	dim3 threadsPerBlock(16,1);
	int *device_blocks;
	hipMalloc(&device_blocks, 1 * sizeof(int));
	hipMemcpy(device_blocks,&host_blocks,sizeof(int)*1,hipMemcpyHostToDevice);
	dim3 numBlocks(host_blocks,1);
	
	readBypassReg<<<numBlocks,threadsPerBlock>>>(devPtrAddr,device_blocks);
	hipError_t cudaerr = hipDeviceSynchronize();
    if (cudaerr != hipSuccess)
        printf("kernel launch failed with error \"%s\".\n",
               hipGetErrorString(cudaerr));
}


