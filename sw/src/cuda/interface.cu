#include "hip/hip_runtime.h"
#include "interface.cuh"
#include "kernel.cuh"
#include "network.cuh"
#include "inference_util.cuh"
#include "sys/time.h"
#include <fstream>
#include <iostream>
#include "tool/log.hpp"

using namespace std;

void socket_sample(param_interface_socket_t param_in){
	socket_context_t* context = get_socket_context(param_in.buffer_addr,param_in.tlb_start_addr,param_in.controller,0);

	int * data;
	size_t total_data_length = size_t(1)*1024*1024*1024;
	hipMalloc(&data,total_data_length);

	sock_addr_t addr;
	addr.ip = param_in.ip;
	addr.port = param_in.port;

	int* socket1;
	hipMalloc(&socket1,sizeof(int));

	int* socket2;
	hipMalloc(&socket2,sizeof(int));

	connection_t* connection1;
	hipMalloc(&connection1,sizeof(connection_t));

	connection_t* connection2;
	hipMalloc(&connection2,sizeof(connection_t));

	hipStream_t stream1,stream2;
	hipEvent_t event1,event2;
	hipStreamCreate(&stream1);
	hipStreamCreate(&stream2);
	hipEventCreate(&event1);
	hipEventCreate(&event2);
	sleep(1);
	cjprint("start user code:\n");

	int verify_data_offset = 5;
	size_t transfer_data_length = size_t(500)*1024*1024;
	param_in.controller->writeReg(165,(unsigned int)(transfer_data_length/64));//count code
	param_in.controller->writeReg(183,(unsigned int)(transfer_data_length/64));
	if(app_type==0){
		//client code
		create_socket<<<1,1,0,stream1>>>(context,socket1);
		compute<<<1,1024,0,stream1>>>(data,total_data_length,verify_data_offset);
		connect<<<1,1,0,stream1>>>(context,socket1,addr);
		socket_send<<<1,1024,0,stream1>>>(context,socket1,data,transfer_data_length);
		//socket_send<<<1,1024,0,stream1>>>(context,socket1,data,transfer_data_length/2);
		//socket_send<<<1,1024,0,stream1>>>(context,socket1,data+(transfer_data_length/2/4),transfer_data_length/2);
		//socket_close<<<1,1,0,stream>>>(context,socket1);
	}else if(app_type==1){
		//server code
		create_socket<<<1,1,0,stream1>>>(context,socket1);
		socket_listen<<<1,1,0,stream1>>>(context,socket1,1235);
		accept<<<1,1,0,stream1>>>(context,socket1,connection1);
		hipEventRecord(event1, stream1);
		hipStreamWaitEvent(stream2, event1,0);
		socket_recv<<<4,1024,0,stream1>>>(context,connection1,data,transfer_data_length);
		socket_recv_ctrl<<<1,16,0,stream2>>>(context,connection1,data,transfer_data_length);
		// hipError_t cudaerr = hipDeviceSynchronize();
		// ErrCheck(cudaerr);
		hipEventRecord(event2, stream2);
		hipStreamWaitEvent(stream1, event2,0);
		verify<<<1,1,0,stream1>>>(data,transfer_data_length,verify_data_offset);

		// hipError_t cudaerr = hipDeviceSynchronize();
		// ErrCheck(cudaerr);
		// sleep(10);

		//socket_close<<<1,1,0,stream>>>(context,connection1);
	}else{
		cjerror("app_type not set!\n");
	}
	sleep(5);
	hipError_t cudaerr = hipPeekAtLastError();
	ErrCheck(cudaerr);


}




void data_mover(param_mover param_mover){
	
	hipError_t err = hipHostRegister(param_mover.write_count_addr0,4,hipHostRegisterIoMemory);
	ErrCheck(err);
	err = hipHostRegister(param_mover.read_count_addr0,4,hipHostRegisterIoMemory);
	ErrCheck(err);

	err = hipHostRegister(param_mover.write_count_addr1,4,hipHostRegisterIoMemory);
	ErrCheck(err);
	err = hipHostRegister(param_mover.read_count_addr1,4,hipHostRegisterIoMemory);
	ErrCheck(err);

	err = hipHostRegister(param_mover.write_count_addr2,4,hipHostRegisterIoMemory);
	ErrCheck(err);
	err = hipHostRegister(param_mover.read_count_addr2,4,hipHostRegisterIoMemory);
	ErrCheck(err);

	err = hipHostRegister(param_mover.write_count_addr3,4,hipHostRegisterIoMemory);
	ErrCheck(err);
	err = hipHostRegister(param_mover.read_count_addr3,4,hipHostRegisterIoMemory);
	ErrCheck(err);

	param_cuda_thread_t param_cuda;

	param_cuda.data_length 		=	param_mover.data_length;
	param_cuda.offset			=	param_mover.offset;
	param_cuda.buffer_pages		=	param_mover.buffer_pages;

	hipHostGetDevicePointer((void **) &(param_cuda.devReadCountAddr0), param_mover.read_count_addr0, 0);
	hipHostGetDevicePointer((void **) &(param_cuda.devReadCountAddr1), param_mover.read_count_addr1, 0);
	hipHostGetDevicePointer((void **) &(param_cuda.devReadCountAddr2), param_mover.read_count_addr2, 0);
	hipHostGetDevicePointer((void **) &(param_cuda.devReadCountAddr3), param_mover.read_count_addr3, 0);

	hipHostGetDevicePointer((void **) &(param_cuda.devWriteCountAddr0), param_mover.write_count_addr0, 0);
	hipHostGetDevicePointer((void **) &(param_cuda.devWriteCountAddr1), param_mover.write_count_addr1, 0);
	hipHostGetDevicePointer((void **) &(param_cuda.devWriteCountAddr2), param_mover.write_count_addr2, 0);
	hipHostGetDevicePointer((void **) &(param_cuda.devWriteCountAddr3), param_mover.write_count_addr3, 0);

	param_cuda.devVAddr0 = param_mover.dev_addr0;
	param_cuda.devVAddr1 = param_mover.dev_addr1;
	param_cuda.devVAddr2 = param_mover.dev_addr2;
	param_cuda.devVAddr3 = param_mover.dev_addr3;

	int hostBlocks=1;
	int threadNum=1024;//8 128
	param_cuda.threadsPerBlock = threadNum;
	param_cuda.blocks = hostBlocks;
	
	
	dim3 threadsPerBlock(threadNum,1);	
	dim3 numBlocks(hostBlocks,1);	  

	//local copy
	// unsigned int * src;
	// hipMalloc(&(src),2000*1024*1024);
	// param_cuda.devVAddr0 = src;

	hipMalloc(&(param_cuda.dstAddr0),param_mover.data_length);
	// hipMalloc(&(param_cuda.dstAddr1),200*1024*1024);
	// hipMalloc(&(param_cuda.dstAddr2),200*1024*1024);
	// hipMalloc(&(param_cuda.dstAddr3),200*1024*1024);
	movThread<<<numBlocks,threadsPerBlock>>>(param_cuda); 
	// hipError_t cudaerr = hipDeviceSynchronize();
	// ErrCheck(cudaerr);
}

void write_bypass(void* addr){
	int peak_clk = 1;
	int device = 0;
	hipError_t e = hipDeviceGetAttribute(&peak_clk, hipDeviceAttributeClockRate, device);
	printf("%d\n",peak_clk);
	hipError_t err = hipHostRegister((void *)addr,1024*1024,hipHostRegisterIoMemory);
	if (err != hipSuccess)
        printf("kernel launch failed with error \"%s\".\n",
               hipGetErrorString(err));
	volatile unsigned int *devPtrAddr;
	hipHostGetDevicePointer((void **) &devPtrAddr, (void *) addr, 0);
	int host_blocks=1;
	dim3 threadsPerBlock(16,1);
	int *device_blocks;
	hipMalloc(&device_blocks, 1 * sizeof(int));
	hipMemcpy(device_blocks,&host_blocks,sizeof(int)*1,hipMemcpyHostToDevice);
	dim3 numBlocks(host_blocks,1);
	
	//writeBypassReg<<<numBlocks,threadsPerBlock>>>(devPtrAddr,device_blocks);
	hipError_t cudaerr = hipDeviceSynchronize();
    if (cudaerr != hipSuccess)
        printf("kernel launch failed with error \"%s\".\n",
               hipGetErrorString(cudaerr));
}
void read_bypass(void* addr){
	int peak_clk = 1;
	int device = 0;
	hipError_t e = hipDeviceGetAttribute(&peak_clk, hipDeviceAttributeClockRate, device);
	printf("%d\n",peak_clk);
	hipError_t err = hipHostRegister((void *)addr,1024*1024,hipHostRegisterIoMemory);
	if (err != hipSuccess)
        printf("kernel launch failed with error \"%s\".\n",
               hipGetErrorString(err));
	volatile unsigned int *devPtrAddr;
	hipHostGetDevicePointer((void **) &devPtrAddr, (void *) addr, 0);
	int host_blocks=1;
	dim3 threadsPerBlock(16,1);
	int *device_blocks;
	hipMalloc(&device_blocks, 1 * sizeof(int));
	hipMemcpy(device_blocks,&host_blocks,sizeof(int)*1,hipMemcpyHostToDevice);
	dim3 numBlocks(host_blocks,1);
	
	//readBypassReg<<<numBlocks,threadsPerBlock>>>(devPtrAddr,device_blocks);
	hipError_t cudaerr = hipDeviceSynchronize();
    if (cudaerr != hipSuccess)
        printf("kernel launch failed with error \"%s\".\n",
               hipGetErrorString(cudaerr));
}


